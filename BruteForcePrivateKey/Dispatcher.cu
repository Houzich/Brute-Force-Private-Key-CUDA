#include "hip/hip_runtime.h"
﻿#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"


const int8_t mapBase58[256] = {
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1, 0, 1, 2, 3, 4, 5, 6,  7, 8,-1,-1,-1,-1,-1,-1,
	-1, 9,10,11,12,13,14,15, 16,-1,17,18,19,20,21,-1,
	22,23,24,25,26,27,28,29, 30,31,32,-1,-1,-1,-1,-1,
	-1,33,34,35,36,37,38,39, 40,41,42,43,-1,44,45,46,
	47,48,49,50,51,52,53,54, 55,56,57,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
	-1,-1,-1,-1,-1,-1,-1,-1, -1,-1,-1,-1,-1,-1,-1,-1,
};

int Generate_Mnemonic_And_Hash(void)
{
	hipError_t cudaStatus = hipSuccess;
	bool find = false;
	size_t round = 0;

	ConfigClass config;
	try {
		parse_gonfig(&config, "config.cfg");
	}
	catch (...) {
		return -1;
	}

	std::string Test = "511111111111111111111111111111111111111111111111111";


	Devices_Info();
	// Choose which GPU to run on, change this on a multi-GPU system.
	uint32_t num_device = 0;
#ifndef GENERATE_INFINITY
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //GENERATE_INFINITY
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = config.cuda_grid * config.cuda_block;


	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
	std::cout << "Seach in pivate key: \"" << config.priv_key << "\"" << std::endl;
	std::cout << "Looking for an address: \"" << config.address << "\"" << std::endl;
	std::string hash160hex;
	data_class *Data = new data_class();
	stride_class* Stride = new stride_class(Data);

	if (config.is_legacy_address)
	{
		tools::decodeAddressBase58(config.address, hash160hex);
		tools::decodeAddressBase58(config.address, (uint8_t*)Data->user_hash160);
	}
	else
	{
		tools::decodeAddressBase32(config.address, hash160hex);
		tools::decodeAddressBase32(config.address, (uint8_t*)Data->user_hash160);
	}
	std::cout << "HASH160: \"" << hash160hex << "\"" << std::endl;


	if (Data->Malloc(config.cuda_grid, config.cuda_block, (uint32_t)config.num_symbols_find) != 0) {
		std::cout << "Error Board->Malloc()!" << std::endl;
		goto Error;
	}
	Data->wif_key = config.priv_key;
	for (int i = 0; i < Data->wif_key.length(); i++)
	{
		Data->user_private_key_wif[i] = mapBase58[Data->wif_key[i]];
	}

	std::cout << "START GENERATE ADDRESSES!" << std::endl;


	if (hipMemcpyToSymbol(HIP_SYMBOL(num_keys_in_packet), &Data->num_keys_round, sizeof(uint32_t), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to num_keys_in_packet failed!");
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(num_sumbols), &Data->num_symbols_find, sizeof(uint32_t), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to num_sumbols failed!");
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(wif_const), Data->user_private_key_wif, 52, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to wif_const failed!");
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(hash_addr), Data->user_hash160, sizeof(uint32_t) * 5, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to hash_addr failed!");
		goto Error;
	}

	while (!find)
	{
		tools::start_time();
		uint8_t random_bytes[52] = { 0 };
		tools::Generate_Bytes_For_PrivKey_Wif_Rundom(Data->num_symbols_find, random_bytes);
		if (hipMemcpyToSymbolAsync(HIP_SYMBOL(random_seed), random_bytes, 52, 0, hipMemcpyHostToDevice, Data->stream1) != hipSuccess)
		{
			fprintf(stderr, "hipMemcpyToSymbol to rundom_seed failed!");
			goto Error;
		}

		if (Stride->start(config.cuda_grid, config.cuda_block) != 0) {
			printf("Error START!!\n");
			goto Error;
		}

		if (Stride->end() != 0) {
			printf("Error END!!\n");
			goto Error;
		}

		if (Data->host.ret->found == 1)
		{
			std::string key_hex = tools::bytesToHexString(Data->host.ret->priv_key, SIZE_PRIVKEY_BYTES);

			tools::Add_Find_Seed_In_File(key_hex, config.address.c_str());
			std::cout << "\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n";
			std::cout << "!!!FOUND KEY " << key_hex << " FOR ADDRESS " << config.address << std::endl;
			std::cout << "!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n";
			break;
		}
		float delay;
		tools::stop_time_and_calc(&delay);
		round++;
		std::cout << "\rSPEED: " << std::setw(8) << std::fixed << tools::formatWithCommas((float)(Data->num_keys_round) / (delay / 1000.0f)) << " KEYS PER SECOND, ROUND: " << round;
	}


	std::cout << "" << std::endl;
	std::cout << "\n\nEND!" << std::endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







