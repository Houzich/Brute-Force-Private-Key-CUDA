﻿#include <stdafx.h>
#include <stdio.h>
#include <stdint.h>


#include "KernelStride.hpp"
#include "Helper.h"
#include <GPU.h>
#include "../Tools/utils.h"

int stride_class::privkey_to_hash160(uint64_t grid, uint64_t block) 
{
	gl_privkey_to_hash_search << <(unsigned int)grid, (unsigned int)block, 0, dt->stream1 >> > (dt->dev.ret);
	return 0;
}




int stride_class::memsetGlobal()
{
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}


int stride_class::start(uint64_t grid, uint64_t block)
{
	if (memsetGlobal() != 0) return -1;
	if (privkey_to_hash160(grid, block) != 0) return -1;

	return 0;
}

int stride_class::end()
{
	hipError_t cudaStatus = hipSuccess;


	//if (DeviceSynchronize("end") != hipSuccess) return -1; //
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}